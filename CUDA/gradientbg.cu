#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"

using namespace optix;

rtDeclareVariable(float3, background_light, , ); // horizon color
rtDeclareVariable(float3, background_dark, , );  // zenith color
rtDeclareVariable(float3, up, , );               // global up vector

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// -----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
  const float t = max(dot(ray.direction, up), 0.0f);
  const float3 result = lerp(background_light, background_dark, t);

  prd_radiance.radiance = result;
  prd_radiance.done = true;
}
